#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "wkv_cuda.h"

#define min(a, b) ((a) < (b) ? (a) : (b))

__global__ void kernel_forward(const int B, const int T, const int C,
                              const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v,
                              float *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    float *__restrict__ const y = _y + _offset;

    float aa = 0, bb = 0, pp = -1e38;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = k[ii];
        const float vv = v[ii];
        const float ww = u + kk;
        const float p = max(pp, ww);
        const float e1 = exp(pp - p);
        const float e2 = exp(ww - p);
        y[ii] = (e1 * aa + e2 * vv) / (e1 * bb + e2);
        const float wkv = w + pp;
        pp = max(wkv, ww);
        const float e1_ = exp(wkv - pp);
        const float e2_ = exp(ww - pp);
        aa = e1_ * aa + e2_ * vv;
        bb = e1_ * bb + e2_;
    }
}

__global__ void kernel_backward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v,
                               const float *__restrict__ const _gy,
                               float *__restrict__ const _gw, float *__restrict__ const _gu, float *__restrict__ const _gk, float *__restrict__ const _gv) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    const float *__restrict__ const gy = _gy + _offset;

    float *__restrict__ const gk = _gk + _offset;
    float *__restrict__ const gv = _gv + _offset;

    float y[Tmax], aa[Tmax], bb[Tmax], pp[Tmax];

    // Forward pass
    float _aa = 0, _bb = 0, _pp = -1e38;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = k[ii];
        const float vv = v[ii];
        const float ww = u + kk;
        const float p = max(_pp, ww);
        const float e1 = exp(_pp - p);
        const float e2 = exp(ww - p);
        y[i] = (e1 * _aa + e2 * vv) / (e1 * _bb + e2);
        const float wkv = w + _pp;
        _pp = max(wkv, ww);
        const float e1_ = exp(wkv - _pp);
        const float e2_ = exp(ww - _pp);
        aa[i] = _aa = e1_ * _aa + e2_ * vv;
        bb[i] = _bb = e1_ * _bb + e2_;
        pp[i] = _pp;
    }

    // Backward pass
    float gaa = 0, gbb = 0, gpp = 0;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const float kk = k[ii];
        const float vv = v[ii];
        const float gyy = gy[ii];
        
        gk[ii] = gyy * gaa;
        gv[ii] = gyy * gbb;
        
        // Additional gradient computations would go here
        // This is a simplified version
    }
    
    _gw[_b * C + _c] = gpp;
    _gu[_b * C + _c] = gaa;
}

void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y) {
    dim3 threadsPerBlock(min(C, 1024));
    dim3 numBlocks((B * C + threadsPerBlock.x - 1) / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}

void cuda_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *gy, float *gw, float *gu, float *gk, float *gv) {
    dim3 threadsPerBlock(min(C, 1024));
    dim3 numBlocks((B * C + threadsPerBlock.x - 1) / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, gy, gw, gu, gk, gv);
}
